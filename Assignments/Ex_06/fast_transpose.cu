//
//  fast_transpose.cu
//
//
//  Created by Laura Balasso on 13/05/2019.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE_DIM 32

/* function that fills an array with random doubles */

void random_doubles(double *p, int n) {
    int i;
    for(i=0; i<n; i++) {
        p[i]= ( (double)rand() * 100 ) / (double)RAND_MAX ;
    }
}


/* function that tests the equality between two martices */

void equality_test(double* M1, double* M2, int N){
    long int i;
    for(i=0; i<N*N; i++){
        if(M1[i]!=M2[i]){
            printf("Error! the two methods produce different results. \n"\
);
            break;
        }
    }
    if(i == N*N) printf("Correct result! \n");
}


/* kernel that implements the fast transpose */

__global__ void fast_transpose(double * M_in , double * M_out, int block_rows){

    __shared__ double tile[TILE_DIM][TILE_DIM]; // allocate the tile in shared memory (one per block)

    int x = blockIdx.x * TILE_DIM + threadIdx.x; // define index
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM; // compute matrix width

    /* each block of threads copies a TILE_DIM x TILE_DIM submatrix in the tile */
    /* some threads handle more than one element since THREADS_PER_BLOCK < TILE_DIM*TILE_DIM */

    for (int j = 0; j < TILE_DIM; j += block_rows){
        tile[threadIdx.y+j][threadIdx.x] = M_in[(y+j) * width + x];
    }
    __syncthreads(); // ensures that all the threads copied the values in the tile

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += block_rows){
        M_out[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
    }
}

/* kernel that implements a naive algorithm for matrix transpose */

__global__ void naive_transpose(double * M_in, double * M_out, int block_rows)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= block_rows){
        M_out[x*width + (y+j)] = M_in[(y+j)*width + x];
    }
}

/* function that runs both the naive and the blocking kernels for a given number of threads per block and matrix size */

void run_kernel(int threads_per_block, int dimx, int dimy){


    /* allocate host matrices */
    int num_bytes = dimx*dimy*sizeof(double);
    double *h_in = (double*)malloc(num_bytes);
    double *h_out_block = (double*)malloc(num_bytes);
    double *h_out_naive = (double*)malloc(num_bytes);

    /* allocate davice matrices */
    double *d_in, *d_out_block, *d_out_naive ;
    hipMalloc(&d_in, num_bytes);
    hipMalloc(&d_out_block, num_bytes);
    hipMalloc(&d_out_naive, num_bytes);
    
    /* fill input matrix with random floats  */
    random_doubles(h_in , dimx*dimy);

    /* copy matrices in device memory */
    hipMemcpy( d_in, h_in, num_bytes, hipMemcpyHostToDevice );
    hipMemcpy( d_out_block, h_out_block, num_bytes, hipMemcpyHostToDevice );
    hipMemcpy( d_out_naive, h_out_naive, num_bytes, hipMemcpyHostToDevice );


    /* set threads and blocks grids */
    int block_rows = threads_per_block/TILE_DIM;
    dim3 grid, block;
    block.x = TILE_DIM;
    block.y = block_rows;
    grid.x = dimx/TILE_DIM;
    grid.y = dimy/TILE_DIM;

    /* cuda events for timing */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    /* run blocking transpose kernel */
    fast_transpose<<< grid, block >>>(d_in, d_out_block, block_rows);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    float bw = 2 * dimx * dimy * sizeof(double) * 1e-6 / ms;
    printf("Fast transpose: \t %lf  \t  %lf \n", ms, bw);

    /* copy the result */
    hipMemcpy( h_out_block, d_out_block, num_bytes, hipMemcpyDeviceToHost );

    hipEventRecord(start);

    /* run naive transpose kernel */
    naive_transpose<<< grid, block >>>(d_in, d_out_naive, block_rows) ;

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms_naive = 0;
    hipEventElapsedTime(&ms_naive, start, stop);
    float bw_naive = 2 * dimx*dimy * sizeof(double) * 1e-6 / ms_naive;
    printf("Naive transpose: \t %lf  \t  %lf \n", ms_naive, bw_naive);

    /* copy result */
    hipMemcpy( h_out_naive, d_out_naive, num_bytes, hipMemcpyDeviceToHost) ;

    equality_test(h_out_naive, h_out_block, dimx);
    
    printf("\n ");

    /* deallocate memory */
    free(h_in); free(h_out_naive); free(h_out_block);
    hipFree(d_in);
    hipFree(d_out_naive);
    hipFree(d_out_block);

    hipEventDestroy(start);
    hipEventDestroy(stop);

}


int main(int argc, char * argv[]){

    int dimx = 8192;
    int dimy = 8192;

    printf(" \t \t \t GPU TIME (ms)     BANDWIDTH (GB/s) ");
    printf("\n\n");
    
    printf("64 threads per block: \n");
    run_kernel(64, dimx, dimy);

    printf("512 threads per block:\n");
    run_kernel(512, dimx, dimy);

    printf("1024 threads per block:\n");
    run_kernel(1024, dimx, dimy);


    return 0;

}










